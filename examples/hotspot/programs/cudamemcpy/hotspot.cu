
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>

#define BLOCK_SIZE 32

#define STR_SIZE 256

/* maximum power density possible (say 300W for a 10mm x 10mm chip)	*/
#define MAX_PD	(3.0e6)
/* required precision in degrees	*/
#define PRECISION	0.001
#define SPEC_HEAT_SI 1.75e6
#define K_SI 100
/* capacitance fitting factor	*/
#define FACTOR_CHIP	0.5

/* chip parameters	*/
float t_chip = 0.0005;
float chip_height = 0.016;
float chip_width = 0.016;
/* ambient temperature, assuming no package at all	*/
float amb_temp = 80.0;

void run(int argc, char** argv);

/* define timer macros */
#define pin_stats_reset()   startCycle()
#define pin_stats_pause(cycles)   stopCycle(cycles)
#define pin_stats_dump(cycles)    printf("timer: %Lu\n", cycles)


#define CUDA_CALL_SAFE(f) \
    do \
    {                                                        \
        hipError_t _cuda_error = f;                         \
        if (_cuda_error != hipSuccess)                      \
        {                                                    \
            fprintf(stderr,  \
                "%s, %d, CUDA ERROR: %s %s\n",  \
                __FILE__,   \
                __LINE__,   \
                hipGetErrorName(_cuda_error),  \
                hipGetErrorString(_cuda_error) \
            ); \
            abort(); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)        

static inline double time_diff(struct timeval tv_start, struct timeval tv_end)
{
    return (double)(tv_end.tv_sec - tv_start.tv_sec) * 1000.0 + (double)(tv_end.tv_usec - tv_start.tv_usec) / 1000.0;
}


/*void writeoutput(float *vect, int grid_rows, int grid_cols, char *file)
{

	int i,j, index=0;
	FILE *fp;
	char str[STR_SIZE];

	if( (fp = fopen(file, "w" )) == 0 )
          printf( "The file was not opened\n" );


	for (i=0; i < grid_rows; i++) 
	 for (j=0; j < grid_cols; j++)
	 {

		 sprintf(str, "%d\t%g\n", index, vect[i*grid_cols+j]);
		 fputs(str,fp);
		 index++;
	 }
		
      fclose(fp);	
}*/

void writeoutput(float *vect, int grid_rows, int grid_cols, char *file)
{
	FILE *fp;

	if ((fp = fopen(file, "wb")) == 0)
    {
        fprintf(stderr, "The file was not opened\n");
        abort();
        exit(EXIT_FAILURE);
    }

    if (fwrite((char *)vect, sizeof(float) * grid_rows * grid_cols, 1, fp) != 1)
    {
        fprintf(stderr, "The file was not written\n");
        abort();
        exit(EXIT_FAILURE);
    }

    fsync(fileno(fp));

	fclose(fp);	
}


/*void readinput(float *vect, int grid_rows, int grid_cols, char *file){

  	int i,j;
	FILE *fp;
	char str[STR_SIZE];
	float val;

	if( (fp  = fopen(file, "r" )) ==0 )
            printf( "The file was not opened\n" );


	for (i=0; i <= grid_rows-1; i++) 
	 for (j=0; j <= grid_cols-1; j++)
	 {
		fgets(str, STR_SIZE, fp);
		if (feof(fp))
			fatal("not enough lines in file");
		//if ((sscanf(str, "%d%f", &index, &val) != 2) || (index != ((i-1)*(grid_cols-2)+j-1)))
		if ((sscanf(str, "%f", &val) != 1))
			fatal("invalid file format");
		vect[i*grid_cols+j] = val;
	}

	fclose(fp);	

}*/

void readinput(float *vect, int grid_rows, int grid_cols, char *file)
{
	FILE *fp;

	if((fp = fopen(file, "rb")) == 0)
    {
        fprintf(stderr, "The file was not opened\n");
        abort();
        exit(EXIT_FAILURE);
    }

    if (fread((char *)vect, sizeof(float) * grid_rows * grid_cols, 1, fp) != 1)
    {
        fprintf(stderr, "The file was not read\n");
        abort();
        exit(EXIT_FAILURE);
    }

	fclose(fp);	
}

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))

__global__ void calculate_temp(long iteration,  //number of iteration
                               float *power,   //power input
                               float *temp_src,    //temperature input/output
                               float *temp_dst,    //temperature input/output
                               long grid_cols,  //Col of grid
                               long grid_rows,  //Row of grid
							   long border_cols,  // border offset 
							   long border_rows,  // border offset
                               float Cap,      //Capacitance
                               float Rx, 
                               float Ry, 
                               float Rz, 
                               float step, 
                               float time_elapsed)
{
	
    __shared__ float temp_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float power_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float temp_t[BLOCK_SIZE][BLOCK_SIZE]; // saving temparary temperature result

	float amb_temp = 80.0;
    float step_div_Cap;
    float Rx_1,Ry_1,Rz_1;
        
	long bx = blockIdx.x;
    long by = blockIdx.y;

	long tx = threadIdx.x;
	long ty = threadIdx.y;
	
	step_div_Cap = step / Cap;
	
	Rx_1 = 1 / Rx;
	Ry_1 = 1 / Ry;
	Rz_1 = 1 / Rz;
	
    // each block finally computes result for a small block
    // after N iterations. 
    // it is the non-overlapping small blocks that cover 
    // all the input data

    // calculate the small block size
	long small_block_rows = BLOCK_SIZE - iteration * 2;//EXPAND_RATE
	long small_block_cols = BLOCK_SIZE - iteration * 2;//EXPAND_RATE

    // calculate the boundary for the block according to 
    // the boundary of its small block
    long blkY = small_block_rows * by - border_rows;
    long blkX = small_block_cols * bx - border_cols;
    long blkYmax = blkY + BLOCK_SIZE - 1;
    long blkXmax = blkX + BLOCK_SIZE - 1;

    // calculate the global thread coordination
	long yidx = blkY + ty;
	long xidx = blkX + tx;

    // load data if it is within the valid input range
	long loadYidx = yidx, loadXidx = xidx;
    long index = grid_cols * loadYidx + loadXidx;
       
	if (IN_RANGE(loadYidx, 0, grid_rows - 1) && IN_RANGE(loadXidx, 0, grid_cols - 1))
    {
        temp_on_cuda[ty][tx] = temp_src[index];  // Load the temperature data from global memory to shared memory
        power_on_cuda[ty][tx] = power[index];    // Load the power data from global memory to shared memory
	}
	__syncthreads();

    // effective range within this block that falls within 
    // the valid range of the input data
    // used to rule out computation outside the boundary.
    long validYmin = (blkY < 0) ? -blkY : 0;
    long validYmax = (blkYmax > grid_rows-1) ? BLOCK_SIZE-1-(blkYmax-grid_rows+1) : BLOCK_SIZE-1;
    long validXmin = (blkX < 0) ? -blkX : 0;
    long validXmax = (blkXmax > grid_cols-1) ? BLOCK_SIZE-1-(blkXmax-grid_cols+1) : BLOCK_SIZE-1;

    long N = ty-1;
    long S = ty+1;
    long W = tx-1;
    long E = tx+1;
    
    N = (N < validYmin) ? validYmin : N;
    S = (S > validYmax) ? validYmax : S;
    W = (W < validXmin) ? validXmin : W;
    E = (E > validXmax) ? validXmax : E;

    bool computed;
    for (long i=0; i<iteration ; i++){ 
        computed = false;
        if( IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) &&  \
              IN_RANGE(ty, i+1, BLOCK_SIZE-i-2) &&  \
              IN_RANGE(tx, validXmin, validXmax) && \
              IN_RANGE(ty, validYmin, validYmax) ) {
              computed = true;
              temp_t[ty][tx] =   temp_on_cuda[ty][tx] + step_div_Cap * (power_on_cuda[ty][tx] + 
                 (temp_on_cuda[S][tx] + temp_on_cuda[N][tx] - 2.0*temp_on_cuda[ty][tx]) * Ry_1 + 
                 (temp_on_cuda[ty][E] + temp_on_cuda[ty][W] - 2.0*temp_on_cuda[ty][tx]) * Rx_1 + 
                 (amb_temp - temp_on_cuda[ty][tx]) * Rz_1);

        }
        __syncthreads();
        if(i==iteration-1)
            break;
        if(computed)	 //Assign the computation range
            temp_on_cuda[ty][tx]= temp_t[ty][tx];
        __syncthreads();
      }

  // update the global memory
  // after the last iteration, only threads coordinated within the 
  // small block perform the calculation and switch on ``computed''
  if (computed){
      temp_dst[index]= temp_t[ty][tx];		
  }
}

/*
   compute N time steps
*/

int compute_tran_temp(float *MatrixPower, float *MatrixTemp[2], long col, long row, \
		long total_iterations, long num_iterations, long blockCols, long blockRows, long borderCols, long borderRows) 
{
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid(blockCols, blockRows);  
	
	float grid_height = chip_height / row;
	float grid_width = chip_width / col;

	float Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * grid_width * grid_height;
	float Rx = grid_width / (2.0 * K_SI * t_chip * grid_height);
	float Ry = grid_height / (2.0 * K_SI * t_chip * grid_width);
	float Rz = t_chip / (K_SI * grid_height * grid_width);

	float max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
	float step = PRECISION / max_slope;
	float t;

    float time_elapsed;
	time_elapsed = 0.001;

    int src = 1, dst = 0;
	
	for (t = 0; t < total_iterations; t += num_iterations) 
    {
        int temp = src;
        src = dst;
        dst = temp;
        calculate_temp<<< dimGrid, dimBlock >>>(MIN(num_iterations, total_iterations-t), MatrixPower, MatrixTemp[src], MatrixTemp[dst], \
            col, row, borderCols, borderRows, Cap, Rx, Ry, Rz, step, time_elapsed);
	}
    CUDA_CALL_SAFE(hipDeviceSynchronize());
    return dst;
}

void usage(int argc, char **argv)
{
	fprintf(stderr, "Usage: %s <grid_rows/grid_cols> <pyramid_height> <sim_time> <temp_file> <power_file> <output_file>\n", argv[0]);
	fprintf(stderr, "\t<grid_rows/grid_cols>  - number of rows/cols in the grid (positive integer)\n");
	fprintf(stderr, "\t<pyramid_height> - pyramid heigh(positive integer)\n");
	fprintf(stderr, "\t<sim_time>   - number of iterations\n");
	fprintf(stderr, "\t<temp_file>  - name of the file containing the initial temperature values of each cell\n");
	fprintf(stderr, "\t<power_file> - name of the file containing the dissipated power values of each cell\n");
	fprintf(stderr, "\t<output_file> - name of the output file\n");
	exit(1);
}

int main(int argc, char** argv)
{
    printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);

    run(argc,argv);

    return EXIT_SUCCESS;
}

void run(int argc, char **argv)
{
    size_t size;
    long grid_rows, grid_cols;
    float *FilesavingTemp, *FilesavingPower, *MatrixOut; 
    char *tfile, *pfile, *ofile;
    
    long total_iterations = 60;
    long pyramid_height = 1; // number of iterations

    struct timeval tv_start, tv_end;
    double kernel_time = 0;       // in ms
    double writefile_time = 0;       // in ms
    double readfile_time = 0;       // in ms
    double d2h_memcpy_time = 0;       // in ms
    double h2d_memcpy_time = 0;       // in ms
	
	if (argc != 7)
		usage(argc, argv);
	if((grid_rows = atol(argv[1]))<=0||
	   (grid_cols = atol(argv[1]))<=0||
       (pyramid_height = atoi(argv[2]))<=0||
       (total_iterations = atoi(argv[3]))<=0)
		usage(argc, argv);
		
	tfile = argv[4];
    pfile = argv[5];
    ofile = argv[6];
	
    size = grid_rows * grid_cols;

    /* --------------- pyramid parameters --------------- */
    # define EXPAND_RATE 2// add one iteration will extend the pyramid base by 2 per each borderline
    long borderCols = (pyramid_height)*EXPAND_RATE/2;
    long borderRows = (pyramid_height)*EXPAND_RATE/2;
    long smallBlockCol = BLOCK_SIZE-(pyramid_height)*EXPAND_RATE;
    long smallBlockRow = BLOCK_SIZE-(pyramid_height)*EXPAND_RATE;
    long blockCols = grid_cols/smallBlockCol+((grid_cols%smallBlockCol==0)?0:1);
    long blockRows = grid_rows/smallBlockRow+((grid_rows%smallBlockRow==0)?0:1);

    FilesavingTemp = (float *)malloc(size * sizeof(float));
    FilesavingPower = (float *)malloc(size * sizeof(float));
    MatrixOut = (float *)calloc(size, sizeof(float));

    if(!FilesavingPower || !FilesavingTemp || !MatrixOut)
    {
        fprintf(stderr, "unable to allocate memory\n");
        abort();
        exit(EXIT_FAILURE);
    }

    printf("pyramidHeight: %d\ngridSize: [%d, %d]\nborder:[%d, %d]\nblockGrid:[%d, %d]\ntargetBlock:[%d, %d]\n",\
        pyramid_height, grid_cols, grid_rows, borderCols, borderRows, blockCols, blockRows, smallBlockCol, smallBlockRow);
	
    gettimeofday(&tv_start, NULL);
    readinput(FilesavingTemp, grid_rows, grid_cols, tfile);
    readinput(FilesavingPower, grid_rows, grid_cols, pfile);
    gettimeofday(&tv_end, NULL);
    readfile_time += time_diff(tv_start, tv_end);

    float *MatrixTemp[2], *MatrixPower;
    CUDA_CALL_SAFE(hipMalloc((void **)&MatrixTemp[0], sizeof(float) * size));
    CUDA_CALL_SAFE(hipMalloc((void **)&MatrixTemp[1], sizeof(float) * size));

    gettimeofday(&tv_start, NULL);
    CUDA_CALL_SAFE(hipMemcpy(MatrixTemp[0], FilesavingTemp, sizeof(float) * size, hipMemcpyHostToDevice));
    gettimeofday(&tv_end, NULL);
    h2d_memcpy_time += time_diff(tv_start, tv_end);

    CUDA_CALL_SAFE(hipMalloc((void **)&MatrixPower, sizeof(float) * size));
    gettimeofday(&tv_start, NULL);
    CUDA_CALL_SAFE(hipMemcpy(MatrixPower, FilesavingPower, sizeof(float) * size, hipMemcpyHostToDevice));
    gettimeofday(&tv_end, NULL);
    h2d_memcpy_time += time_diff(tv_start, tv_end);

    printf("Start computing the transient temperature\n");

    gettimeofday(&tv_start, NULL);
    int ret = compute_tran_temp(MatrixPower,MatrixTemp,grid_cols,grid_rows, \
	    total_iterations,pyramid_height, blockCols, blockRows, borderCols, borderRows);
    gettimeofday(&tv_end, NULL);
    kernel_time += time_diff(tv_start, tv_end);

	printf("Ending simulation\n");
    gettimeofday(&tv_start, NULL);
    CUDA_CALL_SAFE(hipMemcpy(MatrixOut, MatrixTemp[ret], sizeof(float) * size, hipMemcpyDeviceToHost));
    gettimeofday(&tv_end, NULL);
    d2h_memcpy_time += time_diff(tv_start, tv_end);

    gettimeofday(&tv_start, NULL);
    writeoutput(MatrixOut, grid_rows, grid_cols, ofile);
    gettimeofday(&tv_end, NULL);
    writefile_time += time_diff(tv_start, tv_end);

    CUDA_CALL_SAFE(hipFree(MatrixPower));
    CUDA_CALL_SAFE(hipFree(MatrixTemp[0]));
    CUDA_CALL_SAFE(hipFree(MatrixTemp[1]));
    free(MatrixOut);

    printf("==> header: kernel_time (ms),writefile_time (ms),d2h_memcpy_time (ms),readfile_time (ms),h2d_memcpy_time (ms)\n");
    printf("==> data: %f,%f,%f,%f,%f\n", kernel_time, writefile_time, d2h_memcpy_time, readfile_time, h2d_memcpy_time);
}
